/*
  ASSIGNMENT 6: CUDA Intro
  by Martin Rebne Farstad
*/


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

/* Problem size */
#define XSIZE 2560
#define YSIZE 2048

/* Divide the problem into blocks of BLOCKX x BLOCKY threads */
// For CUDA 3.0 and higher, you can use 32 instead of 8 for even more
// parallelism
//#define BLOCKY 8
//#define BLOCKX 8
#define BLOCKY 32
#define BLOCKX 32

#define MAXITER 255

double xleft = -2.01;
double xright = 1;
double yupper, ylower;
double ycenter = 1e-6;
double step;

int host_pixel[XSIZE * YSIZE];
int device_pixel[XSIZE * YSIZE];

typedef struct {
  double real, imag;
} my_complex_t;

#define PIXEL(i, j) ((i) + (j)*XSIZE)

/********** SUBTASK1: Create kernel device_calculate *************************/

/*
  This function is executed on the GPU and therefore the global variables used
  in host_calculate need to be passed as parameters.

  Notice that this kernel is executed for each pixel, while host_calculate
  calculates all pixels
*/
__global__ void device_calculate(int *a, double xleft, double yupper,
                                 double step) {

  int i = blockIdx.x * BLOCKX + threadIdx.x;
  int j = blockIdx.y * BLOCKY + threadIdx.y;
  /* Calculate the number of iterations until divergence for each pixel.
     If divergence never happens, return MAXITER */
  my_complex_t c, z, temp;
  int iter = 0;
  c.real = (xleft + step * i);
  c.imag = (yupper - step * j);
  z = c;
  while (z.real * z.real + z.imag * z.imag < 4.0) {
    temp.real = z.real * z.real - z.imag * z.imag + c.real;
    temp.imag = 2.0 * z.real * z.imag + c.imag;
    z = temp;
    if (++iter == MAXITER)
      break;
  }
  a[PIXEL(i, j)] = iter;
}

/********** SUBTASK1 END *****************************************************/

void host_calculate() {
  for (int j = 0; j < YSIZE; j++) {
    for (int i = 0; i < XSIZE; i++) {
      /* Calculate the number of iterations until divergence for each pixel.
         If divergence never happens, return MAXITER */
      my_complex_t c, z, temp;
      int iter = 0;
      c.real = (xleft + step * i);
      c.imag = (yupper - step * j);
      z = c;
      while (z.real * z.real + z.imag * z.imag < 4.0) {
        temp.real = z.real * z.real - z.imag * z.imag + c.real;
        temp.imag = 2.0 * z.real * z.imag + c.imag;
        z = temp;
        if (++iter == MAXITER)
          break;
      }
      host_pixel[PIXEL(i, j)] = iter;
    }
  }
}

typedef unsigned char uchar;

/* save 24-bits bmp file, buffer must be in bmp format: upside-down */
void savebmp(char *name, uchar *buffer, int x, int y) {
  FILE *f = fopen(name, "wb");
  if (!f) {
    printf("Error writing image to disk.\n");
    return;
  }
  unsigned int size = x * y * 3 + 54;
  uchar header[54] = {'B',
                      'M',
                      size & 255,
                      (size >> 8) & 255,
                      (size >> 16) & 255,
                      size >> 24,
                      0,
                      0,
                      0,
                      0,
                      54,
                      0,
                      0,
                      0,
                      40,
                      0,
                      0,
                      0,
                      x & 255,
                      x >> 8,
                      0,
                      0,
                      y & 255,
                      y >> 8,
                      0,
                      0,
                      1,
                      0,
                      24,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0};
  fwrite(header, 1, 54, f);
  fwrite(buffer, 1, x * y * 3, f);
  fclose(f);
}

/* given iteration number, set a colour */
void fancycolour(uchar *p, int iter) {
  if (iter == MAXITER)
    ;
  else if (iter < 8) {
    p[0] = 128 + iter * 16;
    p[1] = p[2] = 0;
  } else if (iter < 24) {
    p[0] = 255;
    p[1] = p[2] = (iter - 8) * 16;
  } else if (iter < 160) {
    p[0] = p[1] = 255 - (iter - 24) * 2;
    p[2] = 255;
  } else {
    p[0] = p[1] = (iter - 160) * 2;
    p[2] = 255 - (iter - 160) * 2;
  }
}

/*
 * Get system time to microsecond precision (ostensibly, the same as MPI_Wtime),
 * returns time in seconds
 */
double walltime(void) {
  static struct timeval t;
  gettimeofday(&t, NULL);
  return (t.tv_sec + 1e-6 * t.tv_usec);
}

int main(int argc, char **argv) {
  if (argc == 1) {
    puts("Usage: MANDEL n");
    puts("n decides whether image should be written to disk (1=yes, 0=no)");
    return 0;
  }
  double start;
  double hosttime = 0;
  double devicetime = 0;
  double memtime = 0;

  hipDeviceProp_t p;
  hipSetDevice(0);
  hipGetDeviceProperties(&p, 0);
  printf("Device compute capability: %d.%d\n", p.major, p.minor);

  /* Calculate the range in the y-axis such that we preserve the
     aspect ratio */
  step = (xright - xleft) / XSIZE;
  yupper = ycenter + (step * YSIZE) / 2;
  ylower = ycenter - (step * YSIZE) / 2;

  /* Host calculates image */
  start = walltime();
  host_calculate();
  hosttime += walltime() - start;

  /********** SUBTASK2: Set up device memory *******************************/

  /* By using cudaMalloc we allocate memory on the device (GPU). Notice that
   * cudaMemcpy is not required as we do not need to copy any memory from host
   * to device. */
  int *a;
  hipMalloc((void **)&a, XSIZE * YSIZE * sizeof(int));

  /********** SUBTASK2 END *************************************************/

  start = walltime();
  /********** SUBTASK3: Execute the kernel on the device *******************/

  // The amount of blocks in the grid (third dimension defaults to 1)
  dim3 gridBlock(XSIZE / BLOCKX, YSIZE / BLOCKY);
  // The amount of threads in a block (third dimension defaults to 1)
  dim3 threadBlock(BLOCKX, BLOCKY);
  // Execute the kernel on the device
  device_calculate<<<gridBlock, threadBlock>>>(a, xleft, yupper, step);

  /********** SUBTASK3 END *************************************************/

  devicetime += walltime() - start;

  start = walltime();

  /********** SUBTASK4: Transfer the result from device to device_pixel[][]*/

  // Use cudaMemcpy to transfer result from device to host
  hipMemcpy(device_pixel, a, XSIZE * YSIZE * sizeof(int),
             hipMemcpyDeviceToHost);

  /********** SUBTASK4 END *************************************************/

  memtime += walltime() - start;

  /********** SUBTASK5: Free the device memory also ************************/

  // Free the memory allocated on the device
  hipFree(a);

  /********** SUBTASK5 END *************************************************/

  int errors = 0;
  /* check if result is correct */
  for (int i = 0; i < XSIZE; i++) {
    for (int j = 0; j < YSIZE; j++) {
      int diff = host_pixel[PIXEL(i, j)] - device_pixel[PIXEL(i, j)];
      if (diff < 0)
        diff = -diff;
      /* allow +-1 difference */
      if (diff > 1) {
        if (errors < 10)
          printf("Error on pixel %d %d: expected %d, found %d\n", i, j,
                 host_pixel[PIXEL(i, j)], device_pixel[PIXEL(i, j)]);
        else if (errors == 10)
          puts("...");
        errors++;
      }
    }
  }
  if (errors > 0)
    printf("Found %d errors.\n", errors);
  else
    puts("Device calculations are correct.");

  printf("\n");
  printf("Host time:          %7.3f ms\n", hosttime * 1e3);
  printf("Device calculation: %7.3f ms\n", devicetime * 1e3);
  printf("Copy result:        %7.3f ms\n", memtime * 1e3);

  if (strtol(argv[1], NULL, 10) != 0) {
    /* create nice image from iteration counts. take care to create it upside
             down (bmp format) */
    unsigned char *buffer = (unsigned char *)calloc(XSIZE * YSIZE * 3, 1);
    for (int i = 0; i < XSIZE; i++) {
      for (int j = 0; j < YSIZE; j++) {
        int p = ((YSIZE - j - 1) * XSIZE + i) * 3;
        fancycolour(buffer + p, device_pixel[PIXEL(i, j)]);
        // fancycolour(buffer + p, host_pixel[PIXEL(i, j)]);
      }
    }
    /* write image to disk */
    savebmp("mandel1.bmp", buffer, XSIZE, YSIZE);
  }
  return 0;
}
