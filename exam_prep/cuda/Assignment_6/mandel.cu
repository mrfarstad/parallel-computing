
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

/* Problem size */
#define XSIZE 2560
#define YSIZE 2048

/* Divide the problem into blocks of BLOCKX x BLOCKY threads */
#define BLOCKY 32
#define BLOCKX 32

#define MAXITER 255

double xleft = -2.01;
double xright = 1;
double yupper, ylower;
double ycenter = 1e-6;
double step;

int host_pixel[XSIZE * YSIZE];
int device_pixel[XSIZE * YSIZE];
int *dev;

typedef struct {
  double real, imag;
} my_complex_t;

#define PIXEL(i, j) ((i) + (j)*XSIZE)

/********** SUBTASK1: Create kernel device_calculate *************************/

__global__ void device_calculate(int *dev, double xleft, double xright,
                                 double yupper, double ylower, double ycenter,
                                 double step) {
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  /* Calculate the number of iterations until divergence for each pixel.
     If divergence never happens, return MAXITER */
  if (j < YSIZE && i < XSIZE) {
    my_complex_t c, z, temp;
    int iter = 0;
    c.real = (xleft + step * i);
    c.imag = (yupper - step * j);
    z = c;
    while (z.real * z.real + z.imag * z.imag < 4.0) {
      temp.real = z.real * z.real - z.imag * z.imag + c.real;
      temp.imag = 2.0 * z.real * z.imag + c.imag;
      z = temp;
      if (++iter == MAXITER)
        break;
    }
    dev[PIXEL(i, j)] = iter;
  }
}

/********** SUBTASK1 END *****************************************************/

void host_calculate() {
  for (int j = 0; j < YSIZE; j++) {
    for (int i = 0; i < XSIZE; i++) {
      /* Calculate the number of iterations until divergence for each pixel.
         If divergence never happens, return MAXITER */
      my_complex_t c, z, temp;
      int iter = 0;
      c.real = (xleft + step * i);
      c.imag = (yupper - step * j);
      z = c;
      while (z.real * z.real + z.imag * z.imag < 4.0) {
        temp.real = z.real * z.real - z.imag * z.imag + c.real;
        temp.imag = 2.0 * z.real * z.imag + c.imag;
        z = temp;
        if (++iter == MAXITER)
          break;
      }
      host_pixel[PIXEL(i, j)] = iter;
    }
  }
}

typedef unsigned char uchar;

/* save 24-bits bmp file, buffer must be in bmp format: upside-down */
void savebmp(char *name, uchar *buffer, int x, int y) {
  FILE *f = fopen(name, "wb");
  if (!f) {
    printf("Error writing image to disk.\n");
    return;
  }
  unsigned int size = x * y * 3 + 54;
  uchar header[54] = {'B',
                      'M',
                      size & 255,
                      (size >> 8) & 255,
                      (size >> 16) & 255,
                      size >> 24,
                      0,
                      0,
                      0,
                      0,
                      54,
                      0,
                      0,
                      0,
                      40,
                      0,
                      0,
                      0,
                      x & 255,
                      x >> 8,
                      0,
                      0,
                      y & 255,
                      y >> 8,
                      0,
                      0,
                      1,
                      0,
                      24,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0,
                      0};
  fwrite(header, 1, 54, f);
  fwrite(buffer, 1, x * y * 3, f);
  fclose(f);
}

/* given iteration number, set a colour */
void fancycolour(uchar *p, int iter) {
  if (iter == MAXITER)
    ;
  else if (iter < 8) {
    p[0] = 128 + iter * 16;
    p[1] = p[2] = 0;
  } else if (iter < 24) {
    p[0] = 255;
    p[1] = p[2] = (iter - 8) * 16;
  } else if (iter < 160) {
    p[0] = p[1] = 255 - (iter - 24) * 2;
    p[2] = 255;
  } else {
    p[0] = p[1] = (iter - 160) * 2;
    p[2] = 255 - (iter - 160) * 2;
  }
}

/*
 * Get system time to microsecond precision (ostensibly, the same as MPI_Wtime),
 * returns time in seconds
 */
double walltime(void) {
  static struct timeval t;
  gettimeofday(&t, NULL);
  return (t.tv_sec + 1e-6 * t.tv_usec);
}

int main(int argc, char **argv) {
  if (argc == 1) {
    puts("Usage: MANDEL n");
    puts("n decides whether image should be written to disk (1=yes, 0=no)");
    return 0;
  }
  double start;
  double hosttime = 0;
  double devicetime = 0;
  double memtime = 0;

  hipDeviceProp_t p;
  hipSetDevice(0);
  hipGetDeviceProperties(&p, 0);
  printf("Device compute capability: %d.%d\n", p.major, p.minor);

  /* Calculate the range in the y-axis such that we preserve the
     aspect ratio */
  step = (xright - xleft) / XSIZE;
  yupper = ycenter + (step * YSIZE) / 2;
  ylower = ycenter - (step * YSIZE) / 2;

  /* Host calculates image */
  start = walltime();
  host_calculate();
  hosttime += walltime() - start;

  /********** SUBTASK2: Set up device memory *******************************/

  hipMalloc((void **)&dev, XSIZE * YSIZE * sizeof(int));

  /********** SUBTASK2 END *************************************************/

  start = walltime();
  /********** SUBTASK3: Execute the kernel on the device *******************/

  dim3 blockDim(BLOCKX, BLOCKY);
  dim3 gridDim(XSIZE / BLOCKX, YSIZE / BLOCKY);
  device_calculate<<<gridDim, blockDim>>>(dev, xleft, xright, yupper, ylower,
                                          ycenter, step);

  /********** SUBTASK3 END *************************************************/

  devicetime += walltime() - start;

  start = walltime();

  /********** SUBTASK4: Transfer the result from device to device_pixel[][]*/

  hipMemcpy(device_pixel, dev, XSIZE * YSIZE * sizeof(int),
             hipMemcpyDeviceToHost);

  /********** SUBTASK4 END *************************************************/

  memtime += walltime() - start;

  /********** SUBTASK5: Free the device memory also ************************/

  hipFree(dev);

  /********** SUBTASK5 END *************************************************/

  int errors = 0;
  /* check if result is correct */
  for (int i = 0; i < XSIZE; i++) {
    for (int j = 0; j < YSIZE; j++) {
      int diff = host_pixel[PIXEL(i, j)] - device_pixel[PIXEL(i, j)];
      if (diff < 0)
        diff = -diff;
      /* allow +-1 difference */
      if (diff > 1) {
        if (errors < 10)
          printf("Error on pixel %d %d: expected %d, found %d\n", i, j,
                 host_pixel[PIXEL(i, j)], device_pixel[PIXEL(i, j)]);
        else if (errors == 10)
          puts("...");
        errors++;
      }
    }
  }
  if (errors > 0)
    printf("Found %d errors.\n", errors);
  else
    puts("Device calculations are correct.");

  printf("\n");
  printf("Host time:          %7.3f ms\n", hosttime * 1e3);
  printf("Device calculation: %7.3f ms\n", devicetime * 1e3);
  printf("Copy result:        %7.3f ms\n", memtime * 1e3);

  if (strtol(argv[1], NULL, 10) != 0) {
    /* create nice image from iteration counts. take care to create it upside
             down (bmp format) */
    unsigned char *buffer = (unsigned char *)calloc(XSIZE * YSIZE * 3, 1);
    for (int i = 0; i < XSIZE; i++) {
      for (int j = 0; j < YSIZE; j++) {
        int p = ((YSIZE - j - 1) * XSIZE + i) * 3;
        fancycolour(buffer + p, device_pixel[PIXEL(i, j)]);
      }
    }
    /* write image to disk */
    savebmp("mandel1.bmp", buffer, XSIZE, YSIZE);
  }
  return 0;
}
